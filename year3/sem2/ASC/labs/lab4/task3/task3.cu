#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils.h"

/**
 * ~TODO 3~
 * Modify the kernel below such as each element of the
 * array will be now equal to 0 if it is an even number
 * or 1, if it is an odd number
 */
__global__ void kernel_parity_id(int *a, int N) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) {
        int val = a[i] % 2;
        a[i] = val;
    }
}

/**
 * ~TODO 4~
 * Modify the kernel below such as each element will
 * be equal to the BLOCK ID this computation takes
 * place.
 */
__global__ void kernel_block_id(int *a, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        a[i] = blockIdx.x;
    }
}

/**
 * ~TODO 5~
 * Modify the kernel below such as each element will
 * be equal to the THREAD ID this computation takes
 * place.
 */
__global__ void kernel_thread_id(int *a, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        a[i] = threadIdx.x;
    }
}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    /**
     * ~TODO 1~
     * For each device, show some details in the format below,
     * then set as active device the first one (assuming there
     * is at least CUDA-capable device). Pay attention to the
     * type of the fields in the hipDeviceProp_t structure.
     *
     * Device number: <i>
     *      Device name: <name>
     *      Total memory: <mem>
     *      Memory Clock Rate (KHz): <mcr>
     *      Memory Bus Width (bits): <mbw>
     *
     * Hint: look for hipGetDeviceProperties and hipSetDevice in
     * the Cuda Toolkit Documentation.
     */
    // hipDeviceProp_t* prop;
    // for (int i = 0; i < nDevices; ++i) {
    //     hipGetDeviceProperties(prop, i);

    //     puts(prop->name);
    //     printf("%lu\n", prop->totalGlobalMem);
    //     printf("%d\n", prop->memoryClockRate);
    //     printf("%d\n", prop->memoryBusWidth);
    // }

    hipSetDevice(0);

    /**
     * ~TODO 2~
     * With information from example_2.cu, allocate an array with
     * integers (where a[i] = i). Then, modify the three kernels
     * above and execute them using 4 blocks, each with 4 threads.
     *
     * You can use the fill_array(int *a, int n) function (from utils)
     * to fill your array as many times you want.
     *
     *  ~TODO 3~
     * Execute kernel_parity_id kernel and then copy from
     * the device to the host; call hipDeviceSynchronize()
     * after a kernel execution for safety purposes.
     */
    int N = 16;
    int *host_array = 0;
    host_array = (int *) malloc(N * sizeof(int));
    const size_t block_size = 4;
    size_t blocks_no = N / block_size;

    if (N % block_size)
        ++blocks_no;

    fill_array_int(host_array, N);

    int *device_array = 0;
    hipMalloc((void **) &device_array, N * sizeof(int));
    hipMemcpy(device_array, host_array, N * sizeof(int), hipMemcpyHostToDevice);

    kernel_parity_id<<<blocks_no, block_size>>>(device_array, N);
    hipDeviceSynchronize();

    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    check_task_2(3, host_array);


    /**
     * ~TODO 4~
     * Execute kernel_block_id kernel and then copy from
     * the device to the host;
     */
    fill_array_int(host_array, N);
    hipMemcpy(device_array, host_array, N * sizeof(int), hipMemcpyHostToDevice);

    kernel_block_id<<<blocks_no, block_size>>>(device_array, N);
    hipDeviceSynchronize();

    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    check_task_2(4, host_array);

    /**
     * ~TODO 5~
     * Execute kernel_thread_id kernel and then copy from
     * the device to the host;
     */
    fill_array_int(host_array, N);
    hipMemcpy(device_array, host_array, N * sizeof(int), hipMemcpyHostToDevice);

    kernel_thread_id<<<blocks_no, block_size>>>(device_array, N);
    hipDeviceSynchronize();

    hipMemcpy(host_array, device_array, N * sizeof(int), hipMemcpyDeviceToHost);

    check_task_2(5, host_array);

    // TODO 6: Free the memory
    free(host_array);
    // free(host_array_result);
    hipFree(device_array);

    return 0;
}