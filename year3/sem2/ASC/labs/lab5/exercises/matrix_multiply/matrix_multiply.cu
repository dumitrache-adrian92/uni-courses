

#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <algorithm>
#include <iostream>
#include <vector>

#define TILE_WIDTH 16

// TODO 1 - simple matrix multiplication
__global__ void matrix_multiply_simple(float *ma, float *mb, float *mc,
                                       size_t width) {
  // TODO: calculate the row & column index of the element
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // TODO: do dot product between row of ma and column of mb
  if (row >= width || col >= width) {
    return;
  }

  float result = 0;
  for (int k = 0; k < width; ++k) {
    result += ma[row * width + k] * mb[k * width + col];
  }

  // TODO: write result in mc
  mc[row * width + col] = result;
}

// TODO 2 - optimized matrix multiplication
__global__ void matrix_multiply(float *ma, float *mb, float *mc, size_t width) {
  // create shorthand names for threadIdx & blockIdx
  int tx = threadIdx.x, ty = threadIdx.y;
  int bx = blockIdx.x, by = blockIdx.y;

  // TODO: allocate 2D tiles in __shared__ memory
  __shared__ float tile_a[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tile_b[TILE_WIDTH][TILE_WIDTH];

  // TODO: calculate the row & column index of the element
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float result = 0;

  if (row >= width || col >= width) {
    return;
  }

  // loop over the tiles of the input in phases
  for (int t = 0; t < width / TILE_WIDTH; ++t) {
    // TODO: load tiles into __shared__ memory allocated before
    tile_a[ty][tx] = ma[row * width + t * TILE_WIDTH + tx];
    tile_b[ty][tx] = mb[(t * TILE_WIDTH + ty) * width + col];

    // TODO:
    //  wait until all data is loaded before allowing
    //  any thread in this block to continue
    __syncthreads();

    // TODO: do dot product between row of tile from ma and column of tile from
    // mb
    for (int k = 0; k < TILE_WIDTH; ++k) {
      result += tile_a[ty][k] * tile_b[k][tx];
    }

    // TODO:
    //  wait until all data is loaded before allowing
    //  any thread in this block to continue
    __syncthreads();
  }

  // TODO: write result in mc
  mc[row * width + col] = result;
}

int main(void) {
  // create a large workload so we can easily measure the
  // performance difference of both implementations

  // note that n measures the width of the matrix, not the number of total
  // elements
  const size_t n = 1 << 10;
  const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
  const dim3 num_blocks(n / block_size.x, n / block_size.y);

  // generate random input on the host
  std::vector<float> host_a(n * n), host_b(n * n), host_c(n * n);
  for (int i = 0; i < n * n; ++i) {
    host_a[i] = static_cast<float>(rand()) / RAND_MAX;
    host_b[i] = static_cast<float>(rand()) / RAND_MAX;
  }

  // allocate storage for the device
  float *device_a = 0, *device_b = 0, *device_c = 0;
  hipMalloc((void **)&device_a, sizeof(float) * n * n);
  hipMalloc((void **)&device_b, sizeof(float) * n * n);
  hipMalloc((void **)&device_c, sizeof(float) * n * n);

  // copy input to the device
  hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n,
             hipMemcpyHostToDevice);
  hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n,
             hipMemcpyHostToDevice);

  // TODO 3 - measure the time spent in the kernel for simple and optimized
  // implementation

  // TODO: create CUDA events for measuring kernel time
  hipEvent_t launch_begin, launch_end;

  hipEventCreate(&launch_begin);
  hipEventCreate(&launch_end);

  // time many kernel launches and take the average time
  const size_t num_launches = 100;
  float average_simple_time = 0;
  std::cout << "Timing simple implementation...";

  for (int i = 0; i < num_launches; ++i) {
    // TODO: record CUDA event before and after the kernel launch
    hipEventRecord(launch_begin);


    matrix_multiply_simple<<<num_blocks, block_size>>>(device_a, device_b,
                                                       device_c, n);

    // TODO: Wait for launch_end event to complete
    hipEventRecord(launch_end);
    hipEventSynchronize(launch_end);

    // TODO: measure the time spent in the kernel
    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    average_simple_time += time;
  }

  average_simple_time /= num_launches;
  std::cout << " done." << std::endl;

  // now time the optimized kernel

  // time many kernel launches and take the average time
  float average_optimized_time = 0;
  std::cout << "Timing optimized implementation...";
  for (int i = 0; i < num_launches; ++i) {
    // TODO: record CUDA event before and after the kernel launch
    hipEventRecord(launch_begin);

    matrix_multiply<<<num_blocks, block_size>>>(device_a, device_b, device_c,
                                                n);

    // TODO: Wait for launch_end event to complete
    hipEventRecord(launch_end);
    hipEventSynchronize(launch_end);

    // TODO: measure the time spent in the kernel
    float time = 0;
    hipEventElapsedTime(&time, launch_begin, launch_end);

    average_optimized_time += time;
  }

  average_optimized_time /= num_launches;
  std::cout << " done." << std::endl;

  // report the effective throughput of each kernel in GFLOPS
  // the effective throughput is measured as the number of floating point
  // operations performed per second: (one mul + one add) * N^3
  float simple_throughput = static_cast<float>(2 * n * n * n) /
                            (average_simple_time / 1000.0f) / 1000000000.0f;
  float optimized_throughput = static_cast<float>(2 * n * n * n) /
                               (average_optimized_time / 1000.0f) /
                               1000000000.0f;

  std::cout << "Matrix size: " << n << "x" << n << std::endl;
  std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

  std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS"
            << std::endl;
  std::cout << "Throughput of optimized kernel: " << optimized_throughput
            << " GFLOPS" << std::endl;
  std::cout << "Performance improvement: "
            << optimized_throughput / simple_throughput << "x" << std::endl;
  std::cout << std::endl;

  // TODO: destroy the CUDA events
  hipEventDestroy(launch_begin);
  hipEventDestroy(launch_end);

  // deallocate device memory
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);

  return 0;
}
