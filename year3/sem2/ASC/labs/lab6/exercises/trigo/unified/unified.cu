#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../utils.h"

#define NUM_ELEM (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
  float* host_a = 0;
  float* host_b = 0;

  // TODO-unified: Alocare memorie unificata
  hipMallocManaged(&host_a, NUM_ELEM * sizeof(float));
  hipMallocManaged(&host_b, NUM_ELEM * sizeof(float));

  if (host_a == 0 || host_b == 0) {
    printf("[HOST] Couldn't allocate memory\n");
    return 1;
  }

  fill_array_random(host_b, NUM_ELEM);

  kernel_compute<<<NUM_ELEM / 256, 256>>>(host_a, host_b, NUM_ELEM);
  hipDeviceSynchronize();

  // TODO print first partial results
  for (int i = 0; i < 16; i++) {
    printf("host_a[%d] = %f; ", i, host_a[i]);
    printf("host_b[%d] = %f\n", i, host_b[i]);
  }

  // TODO-unified: Dealocare memorie unificata
  hipFree(host_a);
  hipFree(host_b);

  return 0;
}

int main(void) { compute_UnifiedMem(); }
