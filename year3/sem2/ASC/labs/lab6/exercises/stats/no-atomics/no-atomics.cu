
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX_NUM 3

#define INDEX_SUM 0
#define INDEX_MAX 1
#define INDEX_MIN 2

#define NUM_MAX 1024

#define ITEMS_NUM (1024 * 1024)
#define BLOCK_SIZE 256

using namespace std;

// TODO-1 => ./task31
// 1 thread does all compute, no atomic/sync
// thread.0 of block.0 computes everything
__global__ void kernel_no_atomics(int *data, int *results) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    for (int i = 0; i < ITEMS_NUM; i++) {
      results[INDEX_SUM] += data[i];

      results[INDEX_MAX] =
          (data[i] > results[INDEX_MAX]) ? data[i] : results[INDEX_MAX];

      results[INDEX_MIN] =
          (data[i] < results[INDEX_MIN]) ? data[i] : results[INDEX_MIN];
    }
  }
}

int main(void) {
  int expResults[INDEX_NUM];

  int *data = NULL;
  hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
  if (data == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // generate data and expected result
  expResults[INDEX_SUM] = 0;
  expResults[INDEX_MAX] = 0;
  expResults[INDEX_MIN] = NUM_MAX;

  for (int i = 0; i < ITEMS_NUM; i++) {
    // each generated number is lower than NUM_MAX as value
    data[i] = rand() % NUM_MAX;

    expResults[INDEX_SUM] += data[i];

    expResults[INDEX_MAX] =
        (data[i] > expResults[INDEX_MAX]) ? data[i] : expResults[INDEX_MAX];

    expResults[INDEX_MIN] =
        (data[i] < expResults[INDEX_MIN]) ? data[i] : expResults[INDEX_MIN];
  }

  int *results = NULL;
  hipMallocManaged(&results, INDEX_NUM * sizeof(int));
  if (results == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // compute 10 times the results
  for (int i = 0; i < 10; i++) {
    // init
    results[INDEX_SUM] = 0;
    results[INDEX_MAX] = 0;
    results[INDEX_MIN] = NUM_MAX;

    kernel_no_atomics<<<1, 1>>>(data, results);
    hipDeviceSynchronize();
  }

  cout << "SUM: " << results[INDEX_SUM] << endl;
  if (results[INDEX_SUM] != expResults[INDEX_SUM]) {
    cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
  }

  cout << "MAX: " << results[INDEX_MAX] << endl;
  if (results[INDEX_MAX] != expResults[INDEX_MAX]) {
    cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
  }

  cout << "MIN: " << results[INDEX_MIN] << endl;
  if (results[INDEX_MIN] != expResults[INDEX_MIN]) {
    cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
  }

  hipFree(results);
  return 0;
}
