
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX_NUM 3

#define INDEX_SUM 0
#define INDEX_MAX 1
#define INDEX_MIN 2

#define NUM_MAX 1024

#define ITEMS_NUM (1024 * 1024)
#define BLOCK_SIZE 256

using namespace std;

// TODO-2 => ./task32
// ITEMS_NUM / 256 threads, ITEMS_NUM / 256 * 3 atomic calls
// thread.0 of each block does partial compute, than uses atomics to compute
__global__ void kernel_partial_atomics(int *data, int *results) {
  if (threadIdx.x > 0)
		return;

	int start 		= blockIdx.x * BLOCK_SIZE;
	int localRes[INDEX_NUM] = {0};
	localRes[INDEX_MIN] 	= NUM_MAX;

	for (int i = 0; i != BLOCK_SIZE; ++i) {
		localRes[INDEX_SUM] += data[start + i];

		localRes[INDEX_MAX] = (data[start + i] > localRes[INDEX_MAX]) ?
			data[start + i] : localRes[INDEX_MAX];

		localRes[INDEX_MIN] = (data[start + i] < localRes[INDEX_MIN]) ?
			data[start + i] : localRes[INDEX_MIN];
	}

  atomicAdd(&results[INDEX_SUM], localRes[INDEX_SUM]);
  atomicMax(&results[INDEX_MAX], localRes[INDEX_MAX]);
  atomicMin(&results[INDEX_MIN], localRes[INDEX_MIN]);
}

int main(void) {
  int expResults[INDEX_NUM];

  int *data = NULL;
  hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
  if (data == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // generate data and expected result
  expResults[INDEX_SUM] = 0;
  expResults[INDEX_MAX] = 0;
  expResults[INDEX_MIN] = NUM_MAX;

  for (int i = 0; i < ITEMS_NUM; i++) {
    // each generated number is lower than NUM_MAX as value
    data[i] = rand() % NUM_MAX;

    expResults[INDEX_SUM] += data[i];

    expResults[INDEX_MAX] =
        (data[i] > expResults[INDEX_MAX]) ? data[i] : expResults[INDEX_MAX];

    expResults[INDEX_MIN] =
        (data[i] < expResults[INDEX_MIN]) ? data[i] : expResults[INDEX_MIN];
  }

  int *results = NULL;
  hipMallocManaged(&results, INDEX_NUM * sizeof(int));
  if (results == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // compute 10 times the results
  for (int i = 0; i < 10; i++) {
    // init
    results[INDEX_SUM] = 0;
    results[INDEX_MAX] = 0;
    results[INDEX_MIN] = NUM_MAX;
    kernel_partial_atomics<<<ITEMS_NUM / 256, 1>>>(data, results);
    hipDeviceSynchronize();
  }

  cout << "SUM: " << results[INDEX_SUM] << endl;
  if (results[INDEX_SUM] != expResults[INDEX_SUM]) {
    cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
  }

  cout << "MAX: " << results[INDEX_MAX] << endl;
  if (results[INDEX_MAX] != expResults[INDEX_MAX]) {
    cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
  }

  cout << "MIN: " << results[INDEX_MIN] << endl;
  if (results[INDEX_MIN] != expResults[INDEX_MIN]) {
    cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
  }

  hipFree(results);
  return 0;
}
