
#include <hip/hip_runtime.h>
#include <iostream>

#define INDEX_NUM 3

#define INDEX_SUM 0
#define INDEX_MAX 1
#define INDEX_MIN 2

#define NUM_MAX 1024

#define ITEMS_NUM (1024 * 1024)
#define BLOCK_SIZE 256

using namespace std;

// TODO-3 => ./task33
// ITEMS_NUM threads do compute, ITEMS_NUM * 3 atomic calls
// all threads to atomics to compute
__global__ void kernel_full_atomics(int *data, int *results) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // sum
  atomicAdd(&results[INDEX_SUM], data[tid]);

  // max
  atomicMax(&results[INDEX_MAX], data[tid]);

  // min
  atomicMin(&results[INDEX_MIN], data[tid]);
}

int main(void) {
  int expResults[INDEX_NUM];

  int *data = NULL;
  hipMallocManaged(&data, ITEMS_NUM * sizeof(int));
  if (data == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // generate data and expected result
  expResults[INDEX_SUM] = 0;
  expResults[INDEX_MAX] = 0;
  expResults[INDEX_MIN] = NUM_MAX;

  for (int i = 0; i < ITEMS_NUM; i++) {
    // each generated number is lower than NUM_MAX as value
    data[i] = rand() % NUM_MAX;

    expResults[INDEX_SUM] += data[i];

    expResults[INDEX_MAX] =
        (data[i] > expResults[INDEX_MAX]) ? data[i] : expResults[INDEX_MAX];

    expResults[INDEX_MIN] =
        (data[i] < expResults[INDEX_MIN]) ? data[i] : expResults[INDEX_MIN];
  }

  int *results = NULL;
  hipMallocManaged(&results, INDEX_NUM * sizeof(int));
  if (results == 0) {
    cout << "[HOST] Couldn't allocate memory\n";
    return 1;
  }

  // compute 10 times the results
  for (int i = 0; i < 10; i++) {
    // init
    results[INDEX_SUM] = 0;
    results[INDEX_MAX] = 0;
    results[INDEX_MIN] = NUM_MAX;
    kernel_full_atomics<<<ITEMS_NUM / 256, 256>>>(data, results);
    hipDeviceSynchronize();
  }

  cout << "SUM: " << results[INDEX_SUM] << endl;
  if (results[INDEX_SUM] != expResults[INDEX_SUM]) {
    cout << "Failed, SUM should be " << expResults[INDEX_SUM] << endl;
  }

  cout << "MAX: " << results[INDEX_MAX] << endl;
  if (results[INDEX_MAX] != expResults[INDEX_MAX]) {
    cout << "Failed, MAX should be " << expResults[INDEX_MAX] << endl;
  }

  cout << "MIN: " << results[INDEX_MIN] << endl;
  if (results[INDEX_MIN] != expResults[INDEX_MIN]) {
    cout << "Failed, MIN should be " << expResults[INDEX_MIN] << endl;
  }

  hipFree(results);
  return 0;
}
