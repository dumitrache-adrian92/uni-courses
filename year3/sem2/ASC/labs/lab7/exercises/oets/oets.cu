#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUM_ELEMENTS 16

__device__ void swap(int *a, int *b) {
  int temp = *a;
  *a = *b;
  *b = temp;
}

// TODO 2: define parameters
__global__ void oddEvenTranspositionSort(int *array, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid >= n) {
    return;
  }

  for (int i = 0; i < n; i++) {
    if (i % 2 == 0) {  // Even phase
                       // TODO 2: Compare and swap elements if thread id is even
      if (tid % 2 == 0 && tid + 1 < n) {
        if (array[tid] > array[tid + 1]) {
          swap(&array[tid], &array[tid + 1]);
        }
      }
    } else {           // Odd phase
                       // TODO 3: Compare and swap elements if thread id is odd
      if (tid % 2 != 0 && tid + 1 < n) {
        if (array[tid] > array[tid + 1]) {
          swap(&array[tid], &array[tid + 1]);
        }
      }
    }
    // TODO 4: Sync threads
    __syncthreads();
  }
}

void generateData(int *data, int size) {
  srand(time(0));

  for (int i = 0; i < size; i++) {
    data[i] = rand() % 14 + 1;
  }
}

int compare(const void *a, const void *b) {
  return (*(int *)a - *(int *)b);
}

int main() {
  int *array = NULL;
  array = (int *)malloc(NUM_ELEMENTS * sizeof(int));
  generateData(array, NUM_ELEMENTS);

  printf("Original Array: ");
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    printf("%d ", array[i]);
  }
  printf("\n");

  int *d_array;
  // TODO 0: Allocate device array and copy host elements to it
  hipMalloc(&d_array, NUM_ELEMENTS * sizeof(int));
  hipMemcpy(d_array, array, NUM_ELEMENTS * sizeof(int),
             hipMemcpyHostToDevice);

  // TODO 1: Calculate blocks_no and block_size
  int block_size = 16;
  int blocks_no = NUM_ELEMENTS / block_size;

  if (NUM_ELEMENTS % block_size != 0) {
    blocks_no++;
  }

  oddEvenTranspositionSort<<<blocks_no, block_size>>>(d_array, NUM_ELEMENTS);
  hipDeviceSynchronize();

  hipMemcpy(array, d_array, NUM_ELEMENTS * sizeof(int),
             hipMemcpyDeviceToHost);
  hipFree(d_array);

  printf("Sorted Array: ");
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    printf("%d ", array[i]);
  }
  printf("\n");

  int *reference = (int *)malloc(NUM_ELEMENTS * sizeof(int));

  for (int i = 0; i < NUM_ELEMENTS; i++) {
    reference[i] = array[i];
  }

  qsort(reference, NUM_ELEMENTS, sizeof(int), compare);

  for (int i = 0; i < NUM_ELEMENTS; i++) {
    if (array[i] != reference[i]) {
      printf("Test failed\n");
      free(reference);
      free(array);
      return 0;
    }
  }

  free(array);
  return 0;
}
